#include <malloc.h>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

struct lens_t
{
    float x, y, r, d;
};

struct pane_t
{
    int panecount;
    float *panepos = NULL;
    int *lensindex = NULL;
    lens_t *lensdata = NULL;
};

struct result_t
{
    int raydensity;
    int sensordensity;
    int *sensordata = NULL;
};

int causcal(pane_t pane, result_t result)
{
    memset(result.sensordata, 0, (long long)result.sensordensity * result.sensordensity * sizeof(int));
    float rs = 1.0f / result.raydensity;

    for (int i = 0; i < result.raydensity; i++)
    {
        for (int j = 0; j < result.raydensity; j++)
        {

            float x, y, z, kx, ky, invkz;
            y = (i + 0.5f) * rs;
            x = (j + 0.5f) * rs;
            z = 0.0f;
            kx = 0.0f;
            ky = 0.0f;
            invkz = 1.0f;

            for (int k = 0; k < pane.panecount; k++)
            {
                x += (pane.panepos[k] - z) * kx * invkz;
                y += (pane.panepos[k] - z) * ky * invkz;
                if (x <= 0.0f || x >= 1.0f || y <= 0.0f || y >= 1.0f)
                {
                    goto next;
                }
                z = pane.panepos[k];

                float gx = 0.0f, gy = 0.0f;
                float rx, ry;
                int is = pane.lensindex[k];
                int ie = pane.lensindex[k + 1];
                for (int l = is; l < ie; l++)
                {
                    rx = x - pane.lensdata[l].x;
                    ry = y - pane.lensdata[l].y;
                    float r = rx * rx + ry * ry;
                    float invr02 = pane.lensdata[l].r * pane.lensdata[l].r;
                    if (r < invr02 * 0.99999f)
                    {
                        invr02 = 1.0f / invr02;
                        r = r * invr02;
                        r = 4.0f / (1.0f - r);
                        r = -0.5f * exp(4.0f - r) * r * r * invr02 * pane.lensdata[l].d;
                        gx += r * rx;
                        gy += r * ry;
                    }
                }

                kx += gx;
                ky += gy;
                float kp = kx * kx + ky * ky;
                if (kp >= 1.0f)
                {
                    goto next;
                }
                invkz = 1.0f / sqrt(1.0f - kp);
            }

            x += (1.0f - z) * kx * invkz;
            y += (1.0f - z) * ky * invkz;

            if (x > 0.0f && x < 1.0f && y > 0.0f && y < 1.0f)
            {
                int pixelindex = int(x * result.sensordensity) + result.sensordensity * int(y * result.sensordensity);
                result.sensordata[pixelindex]++;
            }

        next:;
        }
    }
    return 0;
}

// 文件读取
int loadconf(const char *fn, pane_t &pane, result_t &result)
{
    size_t n;
    FILE *fi = NULL;
    if (fi = fopen(fn, "rb"))
    {
        n = fread(&result.raydensity, 4, 1, fi);
        n = fread(&result.sensordensity, 4, 1, fi);
        result.sensordata = (int *)malloc((long long)result.sensordensity * result.sensordensity * sizeof(int));

        n = fread(&pane.panecount, 4, 1, fi);
        pane.panepos = (float *)malloc(pane.panecount * sizeof(float));
        n = fread(pane.panepos, 4, pane.panecount, fi);
        pane.lensindex = (int *)malloc((pane.panecount + 1) * sizeof(int));
        n = fread(pane.lensindex, 4, pane.panecount + 1, fi);
        pane.lensdata = (lens_t *)malloc(pane.lensindex[pane.panecount] * sizeof(lens_t));
        n = fread(pane.lensdata, 16, pane.lensindex[pane.panecount], fi);

        fclose(fi);
    }
    printf("%lu\n", n);
    return 0;
}

int main()
{
    result_t result;
    pane_t pane;
    loadconf("./conf.data", pane, result);

    causcal(pane, result);

    FILE *fi = NULL;
    if (fi = fopen("./out.data", "wb"))
    {
        fwrite(result.sensordata, 1, (long long)result.sensordensity * result.sensordensity * sizeof(int), fi);
        fclose(fi);
    }

    return 0;
}